#include "hip/hip_runtime.h"
// CPU Implementation CGBN
#include <iostream>
#include <iomanip>
#include <fstream> 
#include <sstream>
#include <vector>
#include <gmpxx.h>
#include <hip/hip_runtime.h>
#include <gmp.h>
#include "cgbn/cgbn.h"
#include "../utility/cpu_support.h"
#include "../utility/cpu_simple_bn_math.h"
#include "../utility/gpu_support.h"
#include <hip/hip_runtime.h>
#include <chrono>

// IMPORTANT:  DO NOT DEFINE TPI OR BITS BEFORE INCLUDING CGBN
#define TPI 1
#define BITS (36 * 8)

// helpful typedefs for the kernel
typedef cgbn_context_t<TPI>         context_t;
typedef cgbn_env_t<context_t, BITS> env_t;

struct KeyPair {
    cgbn_mem_t<BITS> private_key;
    cgbn_mem_t<BITS> public_key;
};

void printMaxLimits() {
    int maxBlocks, maxThreadsPerBlock;

    // Get the maximum number of blocks per grid
    hipDeviceGetAttribute(&maxBlocks, hipDeviceAttributeMaxGridDimX, 0);
    std::cout << "Maximum blocks per grid: " << maxBlocks << std::endl;

    // Get the maximum number of threads per block
    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0);
    std::cout << "Maximum threads per block: " << maxThreadsPerBlock << std::endl;
}

// // Function to convert cgbn_mem_t limbs to a hexadecimal string
std::string cgbnMemToStringCPU(const cgbn_mem_t<BITS>& value) {
    std::stringstream ss;
    ss << "0x";
    for (int i = BITS / 32 - 1; i >= 0; --i) {
        ss << std::hex << std::setw(8) << std::setfill('0') << value._limbs[i];
    }
    return ss.str();
}

// Function to convert an integer to a hexadecimal string
__host__ __device__ void intToHexStr(uint32_t value, char* output) {
    const char hexChars[] = "0123456789abcdef";
    output[0] = hexChars[(value >> 28) & 0xF];
    output[1] = hexChars[(value >> 24) & 0xF];
    output[2] = hexChars[(value >> 20) & 0xF];
    output[3] = hexChars[(value >> 16) & 0xF];
    output[4] = hexChars[(value >> 12) & 0xF];
    output[5] = hexChars[(value >> 8) & 0xF];
    output[6] = hexChars[(value >> 4) & 0xF];
    output[7] = hexChars[value & 0xF];
    output[8] = '\0';
}

// Function to convert cgbn_mem_t limbs to a hexadecimal string
__host__ __device__ void cgbnMemToStringGPU(const cgbn_mem_t<BITS>& value, char* output) {
    int index = 0;
    for (int i = BITS / 32 - 1; i >= 0; --i) {
        intToHexStr(value._limbs[i], output + index);
        index += 8;
    }
}

// Helper function to perform addition or subtraction
void performOperation(cgbn_mem_t<BITS>& publicKey, cgbn_mem_t<BITS>& operand, char operation) {

    if (operation == 'A') {
        add_words(publicKey._limbs, publicKey._limbs, operand._limbs, BITS/32);
        // publicKey += operand;
    } else if (operation == 'S') {
        // publicKey -= operand;
        sub_words(publicKey._limbs, publicKey._limbs, operand._limbs, BITS/32);
    }
}

// Helper function to read key pairs from a file
std::vector<KeyPair> readKeyPairs(const std::string& filename) {
    std::vector<KeyPair> keyPairs;
    std::ifstream file(filename);

    if (file.is_open()) {
        std::string line;
        while (std::getline(file, line)) {
            std::istringstream iss(line);
            std::string token;
            KeyPair keyPair;

            // Read Private_k label
            iss >> token;  // Read the label
            if (token != "Private_k:") {
                // Handle error or skip line
                continue;
            }

            // Read private key (in hexadecimal format)
            set_words(keyPair.private_key._limbs, "", BITS / 32);
            iss >> token;
            set_words(keyPair.private_key._limbs, token.c_str(), BITS / 32);

            // Read Public_k label
            iss >> token;  // Read the label
            if (token != "Public_k:") {
                // Handle error or skip line
                continue;
            }

            // Read public key (in hexadecimal format)
            set_words(keyPair.public_key._limbs, "", BITS / 32);
            iss >> token;
            set_words(keyPair.public_key._limbs, token.c_str(), BITS / 32);

            keyPairs.push_back(keyPair);
        }
        file.close();
    }

    return keyPairs;
}


// Helper function to save the matched public key, iteration count, and result to a file
void saveMatchToFile(const std::string& matchFile, const std::string& iteration, const std::string& publicKey) {
    std::ofstream file(matchFile, std::ios::app);
    if (file.is_open()) {
        file << "Iteration Count: " << iteration << std::endl;
        file << "Matched Public Key: [" << publicKey << "]" << std::endl;
        file.close();
    }
}

// GPU kernel for comparing results in parallel
__global__ void kernel_compare(cgbn_error_report_t *report, cgbn_mem_t<BITS> publicKey, KeyPair* botKeyPairs, cgbn_mem_t<BITS>* matchedKey, uint32_t numResults, bool* matchFound, uint32_t iterInstanceCount, int* iterCount) {
    int instance = (blockIdx.x * blockDim.x + threadIdx.x )/ TPI;

    if ((instance < numResults) && !(*matchFound)) 
    {
        cgbn_mem_t<BITS>& botPublicKey = botKeyPairs[instance].public_key;

        context_t      bn_context(cgbn_report_monitor, report, instance);   // construct a context
        env_t          bn_env(bn_context.env<env_t>());                     // construct an environment for 1024-bit math
        env_t::cgbn_t  a, b;                                             // define a, b, r as 1024-bit bignums

        cgbn_load(bn_env, a, &publicKey);      // load my instance's a value
        cgbn_load(bn_env, b, &botPublicKey);      // load my instance's b value

        int comparisonResult = cgbn_equals(bn_env, a, b);

        if (comparisonResult) {
            *matchFound = true;
            *iterCount = iterInstanceCount;
            cgbn_store(bn_env, matchedKey, a);   // store r into sum
        }
    }
}


bool checkCudaAvailability() {
    int deviceCount;
    hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);

    if (cudaStatus != hipSuccess) {
        std::cerr << "Error getting device count: " << hipGetErrorString(cudaStatus) << std::endl;
        return false;
    }

    if (deviceCount == 0) {
        std::cerr << "No CUDA-enabled GPU device found." << std::endl;
        return false;
    }

    std::cout << "Found " << deviceCount << " CUDA-enabled GPU device(s)." << std::endl;

    // You can also print more information about each device if needed
    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        std::cout << "Device " << i << ": " << deviceProp.name << std::endl;
        // Print more properties if needed
    }

    return true;
}

__global__ void kernel_iterate(cgbn_error_report_t *report, cgbn_mem_t<BITS>* publicKeys, KeyPair* botKeyPairs, cgbn_mem_t<BITS>* matchedKey, char operationType, const cgbn_mem_t<BITS>* operands, uint32_t numIterations, int numResults, bool* matchFound, int* iterCount) {
    uint32_t instance = (blockIdx.x * blockDim.x + threadIdx.x);
    // cgbn_mem_t<BITS> iterationValue;
    // iterationValue._limbs[0] = instance;
    cgbn_mem_t<BITS> alteredKey;

    if (((instance < numIterations) && !(*matchFound))) {
        cgbn_mem_t<BITS> publicKey = publicKeys[0];
        cgbn_mem_t<BITS> operand = operands[0];

        typedef cgbn_context_t<1>         context_single_t;
        typedef cgbn_env_t<context_single_t, BITS> env_single_t;
        context_single_t      bn_context(cgbn_report_monitor, report, instance);   // construct a context
        env_single_t          bn_env(bn_context.env<env_single_t>());                     // construct an environment for 1024-bit math
        env_single_t::cgbn_t  pKey, op, r, iter;                                             // define a, b, r as 1024-bit bignums
        // env_single_t::cgbn_t rMul;

        cgbn_load(bn_env, pKey, &publicKey);      // load my instance's a value
        cgbn_load(bn_env, op, &operand);      // load my instance's b value
        // cgbn_load(bn_env, iter, &iterationValue);      // load my instance's b value

        cgbn_mul_ui32(bn_env, iter, op, instance);
        // Generate a new key by adding (operand * iteration) to the public key

        // cgbn_mul(bn_env, r, iter, op);

        if (operationType == 'A') 
        {
            cgbn_add(bn_env, r, pKey, iter);
        } 
        else if (operationType == 'S') 
        {
            cgbn_sub(bn_env, r, pKey, iter);     
        }    

        cgbn_store(bn_env, &alteredKey, r);   

        // Now, launch the compare kernel to check for matches
        // Launch the GPU kernel
        uint32_t block_size = 512;
        uint32_t num_blocks = (numResults + block_size - 1) / block_size;
        // char pString[100];
        // cgbnMemToStringGPU(alteredKey, pString);
        // printf("0x%s\n", pString);
        kernel_compare<<<num_blocks, block_size * TPI>>>(report, alteredKey, botKeyPairs, matchedKey, numResults, matchFound, instance, iterCount);
    }
}

// Function to perform GPU comparison
bool performGPUComparison(cgbn_mem_t<BITS>* h_publicKey, const std::vector<KeyPair>& botKeyPairs, char operationType, cgbn_mem_t<BITS>* h_operand, uint32_t numIterations, const std::string matchFile) {
    bool matchFound = false;  // Variable to control the loop
    int iterCount = 0;  // Variable to control the loop
    cgbn_mem_t<BITS> matchedKey;

    cgbn_mem_t<BITS>* d_publicKey;
    cgbn_mem_t<BITS>* d_operand;
    cgbn_mem_t<BITS>* d_matchedKey;
    KeyPair* d_botKeyPairs;
    bool* d_matchFound;
    int* d_iterCount;
    cgbn_error_report_t *report;

    // Allocate memory on the GPU
    CUDA_CHECK(hipMalloc((void**)&d_publicKey, sizeof(cgbn_mem_t<BITS>)));
    CUDA_CHECK(hipMalloc((void**)&d_operand, sizeof(cgbn_mem_t<BITS>)));
    CUDA_CHECK(hipMalloc((void**)&d_matchedKey, sizeof(cgbn_mem_t<BITS>)));
    CUDA_CHECK(hipMalloc((void**)&d_botKeyPairs, botKeyPairs.size() * sizeof(KeyPair)));
    CUDA_CHECK(hipMalloc((void**)&d_matchFound, sizeof(bool)));
    CUDA_CHECK(hipMalloc((void**)&d_iterCount, sizeof(int)));

    // Copy data to the GPU
    CUDA_CHECK(hipMemcpy(d_publicKey, h_publicKey, sizeof(cgbn_mem_t<BITS>), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_operand, h_operand, sizeof(cgbn_mem_t<BITS>), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_botKeyPairs, botKeyPairs.data(), botKeyPairs.size() * sizeof(KeyPair), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_matchFound, &matchFound, sizeof(bool), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_iterCount, &iterCount, sizeof(int), hipMemcpyHostToDevice));

    // create a cgbn_error_report for CGBN to report back errors
    CUDA_CHECK(cgbn_error_report_alloc(&report)); 

    uint32_t numResults = botKeyPairs.size();

    // Before the kernel launch
    hipError_t cudaStatus;

    // Launch the GPU kernel
    uint32_t block_size = 512;
    uint32_t num_blocks = (numIterations + block_size - 1U) / block_size;
    kernel_iterate<<<num_blocks, block_size>>>(report, d_publicKey, d_botKeyPairs, d_matchedKey, operationType, d_operand, numIterations, numResults, d_matchFound, d_iterCount);

    // Check for kernel launch errors
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "Kernel launch error: " << hipGetErrorString(cudaStatus) << std::endl;
        // Additional error handling or debugging steps can be added here
    }
    // Wait for the kernel to finish
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy back the result
    CUDA_CHECK(hipMemcpy(&matchFound, d_matchFound, sizeof(bool), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(&iterCount, d_iterCount, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(&matchedKey, d_matchedKey, sizeof(cgbn_mem_t<BITS>), hipMemcpyDeviceToHost));

    // Free GPU memory
    CUDA_CHECK(hipFree(d_publicKey));
    CUDA_CHECK(hipFree(d_botKeyPairs));
    CUDA_CHECK(hipFree(d_matchFound));
    CUDA_CHECK(hipFree(d_iterCount));

    if (matchFound) {
        std::cout << std::endl << "Match found at Iteration " << iterCount << std::endl;
        saveMatchToFile(matchFile, std::to_string(iterCount), cgbnMemToStringCPU(matchedKey));
    }

    return matchFound;
}

int main(int argc, char* argv[]) {
    
    if (argc != 6) {
        std::cerr << "Usage: " << argv[0] << " <public_key> <operation_value> <operation_type(A/S)> <num_iterations> <match_file>\n";
        return 1;
    }

    if (checkCudaAvailability()) {
        // Perform GPU-related tasks here
        std::cout << "GPU is available. Proceed with GPU-related tasks." << std::endl;
        printMaxLimits();
    } else {
        // Perform CPU-only tasks here
        std::cout << "No GPU available. Proceed with CPU-only tasks." << std::endl;
    }

    // Read the public key as a string and convert to cgbn_mem_t
    cgbn_mem_t<BITS> publicKey;
    set_words(publicKey._limbs, argv[1], BITS / 32);

    // Read the operand as a string and convert to cgbn_mem_t
    cgbn_mem_t<BITS> operand;
    set_words(operand._limbs, argv[2], BITS / 32);

    char operationType = argv[3][0];
    
    cgbn_mem_t<BITS> numIterations;
    set_words(numIterations._limbs, argv[4], BITS / 32);
    uint64_t numIterationsInt = 0;
    memcpy(&numIterationsInt, numIterations._limbs, sizeof(uint64_t));

    const std::string matchFile = argv[5];

    std::cout << "Entered public key: " << argv[1] << std::endl;
    std::cout << "Entered operand: " << argv[2] << std::endl << std::endl;
    std::cout << "Entered Number of Iterations: " << cgbnMemToStringCPU(numIterations) << std::endl << std::endl;
    auto start_time = std::chrono::high_resolution_clock::now();  // Record the start time

    // Read key pairs from bot.txt
    std::vector<KeyPair> botKeyPairs = readKeyPairs("bot.txt");

    // performOperation(publicKey, operand, operationType);

    // Check if the result matches any public keys in bot.txt
    bool matchResult = performGPUComparison(&publicKey, botKeyPairs, operationType, &operand, numIterations._limbs[0], matchFile);

    if (!matchResult){
        std::cout << std::endl << "No Match found " << std::endl;
    }

    auto end_time = std::chrono::high_resolution_clock::now();  // Record the end time
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);  // Calculate the duration in milliseconds
    std::cout << std::endl << "Program duration: " << duration.count() << " milliseconds" << std::endl;
    std::cout << std::endl << "Program duration: " << duration.count() / 1000.0 << " seconds" << std::endl;



    return 0;
}
