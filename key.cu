#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <gmpxx.h>

// CUDA kernel for performing addition or subtraction on the GPU
__global__ void performOperationKernel(mpz_class* publicKey, const mpz_class operand, char operation, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        if (operation == 'A') {
            publicKey[tid] += operand;
        } else if (operation == 'S') {
            publicKey[tid] -= operand;
        }
    }
}

// Helper function to perform addition or subtraction on the GPU
void performOperationGPU(mpz_class* publicKey, const mpz_class operand, char operation, int size) {
    mpz_class* d_publicKey;
    hipMalloc((void**)&d_publicKey, size * sizeof(mpz_class));
    hipMemcpy(d_publicKey, publicKey, size * sizeof(mpz_class), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;

    performOperationKernel<<<gridSize, blockSize>>>(d_publicKey, operand, operation, size);

    hipMemcpy(publicKey, d_publicKey, size * sizeof(mpz_class), hipMemcpyDeviceToHost);
    hipFree(d_publicKey);
}

struct KeyPair {
    mpz_class private_key;
    mpz_class public_key;
};

// Helper function to read key pairs from a file
std::vector<KeyPair> readKeyPairs(const std::string& filename) {
    std::vector<KeyPair> keyPairs;
    std::ifstream file(filename);

    if (file.is_open()) {
        std::string line;
        while (std::getline(file, line)) {
            std::istringstream iss(line);
            std::string token;
            KeyPair keyPair;

            // Read Private_k label
            iss >> token;  // Read the label
            if (token != "Private_k:") {
                // Handle error or skip line
                continue;
            }

            // Read private key (in hexadecimal format)
            iss >> std::hex >> keyPair.private_key;

            // Read Public_k label
            iss >> token;  // Read the label
            if (token != "Public_k:") {
                // Handle error or skip line
                continue;
            }

            // Read public key (in hexadecimal format)
            iss >> std::hex >> keyPair.public_key;

            keyPairs.push_back(keyPair);
        }
        file.close();
    }

    return keyPairs;
}

// Helper function to save the matched public key, iteration count, and result to a file
void saveMatchToFile(const std::string& matchFile, int iteration, const std::string& publicKey, const std::string& result) {
    std::ofstream file(matchFile, std::ios::app);
    if (file.is_open()) {
        file << "Iteration Count: " << iteration << std::endl;
        file << "Matched Public Key: [" << publicKey << "]" << std::endl;
        file.close();
    }
}

int main(int argc, char* argv[]) {
    if (argc != 6) {
        std::cerr << "Usage: " << argv[0] << " <public_key> <operation_value> <operation_type(A/S)> <num_iterations> <match_file>\n";
        return 1;
    }

    mpz_class publicKey(argv[1], 16);
    mpz_class operand(argv[2]);
    char operationType = argv[3][0];
    int numIterations = std::stoi(argv[4]);
    const std::string matchFile = argv[5];

    std::cout << "Entered public key: " << publicKey.get_str(16) << std::endl;

    // Read key pairs from bot.txt
    std::vector<KeyPair> botKeyPairs = readKeyPairs("bot.txt");

    bool matchFound = false;

    for (int iteration = 1; iteration <= numIterations && !matchFound; ++iteration) {
        std::cout << "Iteration Count: " << iteration << std::endl;

        // Perform the specified operation on GPU
        performOperationGPU(&publicKey, operand, operationType, 1);

        // Display the result
        std::cout << "Result: " << publicKey.get_str(16) << std::endl;

        // Check if the result matches any public keys in bot.txt
        for (const KeyPair& botKeyPair : botKeyPairs) {
            if (publicKey == botKeyPair.public_key) {
                // Match found, save the information to matchFile
                saveMatchToFile(matchFile, iteration, botKeyPair.public_key.get_str(16), publicKey.get_str(16));
                std::cout << std::endl << "Match found at Iteration " << iteration << std::endl;

                // Set the flag to true to exit both loops
                matchFound = true;
                break;
            }
        }
    }

    return 0;
}
